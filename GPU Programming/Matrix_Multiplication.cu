#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include"../common/book.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include<math.h>

#define N 3

/*Defining kernel function matrix multiplication which takes 3 arguments A,B,C these are matrix of 3*3 A & B are input matrix and matrix C is the product of A & B */

__global__ void matrixMult(float *A, float *B, float *C) 
{
	float Sum_Matrix = 0;
	int row = threadIdx.y + blockDim.y * blockIdx.y; // row represents the indices of matrix A
	int col = threadIdx.x + blockDim.x * blockIdx.x; // col represents the indices of matrix B 
	//printf("ThreadIdx.x : %d\tblockDim.x : %d\tblockIdx.x : %d\tThreadIdx.y : %d\tblockDim.y : %d\tblockIdx.y :%d\t\n", threadIdx.x, blockDim.x, blockIdx.x, threadIdx.y, blockDim.y, blockIdx.y);
	
	int index = row * N + col;
	//printf("Index = %d\n",index);
	if (row < N && col < N)
	{
		// each thread computes one element of the block sub- matrix
		for (int i = 0; i < N; ++i) 
		{
			Sum_Matrix += A[i + row * N] * B[col + i * N];
		}
	}
	C[index] = Sum_Matrix;
}


int main() 
{
	/*Declaring matrix A & B of size N*N */
	float A[N][N], B[N][N], C[N][N]; 
	int i, j; // Declaring i for row and j for column of matrix

	/* creating three 2D arrays  */
	float *dev_a, *dev_b, *dev_c; 

	/*--------Taking user input for matrix A elements----------*/
	printf("\n Enter elements of first matrix A of size %d * %d\n", N, N);
	for(i = 0; i<N; i++) // i is representing row of matrix A
	{
		for(j = 0; j<N; j++) // j is representing column of matrix A
		{
			printf("Enter the element A[%d][%d] : ", i, j);
			scanf("%f", &A[i][j]);
		}
	}

	/*--------Taking user input for matrix B elements----------*/
	printf("\n Enter elements of second matrix B of size %d * %d\n", N, N);
	for(i = 0; i<N; i++) // i is representing row of matrix B
	{
		for(j = 0; j<N; j++) // j is representing column of matrix B
		{
			printf("Enter the element B[%d][%d] : ", i, j);
			scanf("%f", &B[i][j]);
		}
	
	}
	/*--------Allocating memory in GPU by using hipMalloc----------*/

	hipMalloc((void**)&dev_a, (N*N) * sizeof(float));
	hipMalloc((void**)&dev_b, (N*N) * sizeof(float));
	hipMalloc((void**)&dev_c, (N*N) * sizeof(float));

	/*--------Copying elements of 2D array A, B from host(CPU) to device(GPU) by using hipMemcpy----------*/

	hipMemcpy(dev_a, A, (N*N) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, B, (N*N) * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, C, (N*N) * sizeof(float), hipMemcpyHostToDevice);

	/*---------Calling kernel function-------------*/
	dim3 blocksPerGrid(1, 1); // Number of blocks is 1
	dim3 threadsPerBlock(N, N); // Number of threadsPerBlock is 9 (3*3)
	matrixMult<<< blocksPerGrid,threadsPerBlock >>>(dev_a, dev_b, dev_c); // Calling kernel function with 1 block and 9 threads per block
	hipDeviceSynchronize(); // synchronizing CPU with GPU

	/*-------- After the GPU kernel function executes it copies the 2D array back from GPU to CPU ----------------*/

	hipMemcpy(C, dev_c, (N*N) * sizeof(float), hipMemcpyDeviceToHost);

	/*-----------------printing the product of two matrix A & B --------------------*/

	printf("\n");
	printf("Product of two matrix A and B is :\n\n ");

	for(i = 0; i<N; i++)
	{
		for(j = 0; j<N; j++)
		{
			printf("%.2f\t\t", C[i][j]);
		}
		printf("\n");
	}

	// Free the memory allocated in GPU
	hipFree(dev_a); 
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}
